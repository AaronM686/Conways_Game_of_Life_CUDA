#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// CUDA implementation of "Conway's Game of Life" cellular automaton.
//   https://en.wikipedia.org/wiki/Conway%27s_Game_of_Life
//
// This is a coding skills demonstration created by Aaron Mosher.
// https://github.com/AaronM686
//
// Makefile and boilerplate support code is based on Nvidia samples "Template"
// You will need the Samples directory to compile this, since
// I rely on several helper-functions they provide to streamline the code.
// 
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>  // from the "/common/inc/" folder of the Nvidia CUDA samples.
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C"
void computeTick(float *reference, float *idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void
testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[]; // AyM: The size of this is determiend by the 3rd paramter of the Kernel invocation.

    // access thread id
    const unsigned int tid = threadIdx.x; // AyM: Need to update this for a 2-dimensional thread block.
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x; // AyM: Need to update this for a 2-dimensional thread block.

    // read in input data from global memory
    sdata[tid] = g_idata[tid];
    __syncthreads(); // a barrier at which all threads in the Block must wait before any is allowed to proceed

    // perform some computations
    // TODO
    __syncthreads(); // a barrier at which all threads in the block must wait before any is allowed to proceed

    // write data to global memory
    g_odata[tid] = sdata[tid];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
// AyM: the main Run function for the iteration of "Conway's game of life"
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);
    // AyM NOTE: This requires the "hip/hip_runtime_api.h" from the "/common/inc/" folder of the Nvidia CUDA samples.

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);
    sdkStartTimer(&timer);

    unsigned int num_threads = 32; // AyM: Need to update this for a 2-dimensional thread block.
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    checkCudaErrors(hipMalloc((void **) &d_idata, mem_size));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(d_idata, h_idata, mem_size,
                               hipMemcpyHostToDevice));

    // allocate device memory for result
    float *d_odata;
    checkCudaErrors(hipMalloc((void **) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel. AyM Note: the 3rd parameter is Shared Memory allocation size for the CUDA block.
    // Given this only runs on Maxwell or higher architectures, do I really need the Shared Memory anymore?
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);

    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,
                               hipMemcpyDeviceToHost));

    sdkStopTimer(&timer);
    printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    // compute reference solution
    float *reference = (float *) malloc(mem_size);
    computeTick(reference, h_idata, num_threads);

    // check result
    if (checkCmdLineFlag(argc, (const char **) argv, "regression"))
    {
        // write file for regression test
        sdkWriteFile("./data/regression.dat", h_odata, num_threads, 0.0f, false);
    }
    else
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected solution
        bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);
    }

    // cleanup memory
    free(h_idata);
    free(h_odata);
    free(reference);
    checkCudaErrors(hipFree(d_idata));
    checkCudaErrors(hipFree(d_odata));

    exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
